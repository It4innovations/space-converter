#include "hip/hip_runtime.h"
/*
 * Copyright(C) 2023-2025 IT4Innovations National Supercomputing Center, VSB - Technical University of Ostrava
 *
 * This program is free software : you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *
 */

#include "cudakdtree_tool.h"

#include "cukd/cukd-math.h"
#include "cukd/traverse-stack-free.h"
#include "cukd/knn.h"
#include <mpi.h>
#include <stdexcept>
#include <hip/hip_runtime.h>

#define CUKD_MPI_CALL(fctCall)                                          \
  { int rc = MPI_##fctCall;                                             \
    if (rc != MPI_SUCCESS)                                              \
      throw std::runtime_error(std::string(__PRETTY_FUNCTION__)+#fctCall); }

using cukd::divRoundUp;

#include <climits>  // For INT_MAX
// Use a macro for max bytes per transfer (less than INT_MAX to be safe)
#define MAX_MPI_BYTES (INT_MAX - 1024)

#include "utility/dense_utility.h"

namespace utility {
	namespace cudakdtree {

        __global__ void runQuery(float3* tree, int N,
            uint64_t* candidateLists, int k, float maxRadius,
            float3* queries, int numQueries,
            int round)
        {
            size_t tid = threadIdx.x + (size_t)blockIdx.x * blockDim.x;
            if (tid >= numQueries) return;

            float3 qp = queries[tid];
            cukd::FlexHeapCandidateList cl(candidateLists + (size_t)k * tid, k,
                round == 0 ? maxRadius : -1.f);
            cukd::stackFree::knn(cl, qp, tree, N);
        }

        __global__ void extractFinalResult(
            float* d_radius_particles,
            float* d_rho_particles,
            float* d_mass_particles,
            int numPoints,
            int k,
            uint64_t* candidateLists,
            common::SpaceData::DenseType rho_kernel
            )
        {
            size_t tid = threadIdx.x + (size_t)blockIdx.x * blockDim.x;
            if (tid >= numPoints) return;

            cukd::FlexHeapCandidateList cl(candidateLists + (size_t)k * tid, k, -1.f);
            float result = cl.returnValue();
            if (!isinf(result)) {
                result = sqrtf(result);
                d_radius_particles[tid] = result;

                //utility::dense::sph_kernel::WendlandC6 kernel_wendland;

                double h_inv = 1.0 / d_radius_particles[tid];

                for (int i = 0; i < k; i++) {
                    float result_i = cukd::uint_as_float(cl.entry[i] >> 32); //cl.get_dist2(i);
                    if (!isinf(result_i) && !isinf(result)) {
                        result_i = sqrtf(result_i);

                        // SPH density estimate by SPHtoGrid
                        d_rho_particles[tid] += d_mass_particles[tid] * utility::dense::sph_kernel::W(rho_kernel, result_i * h_inv, h_inv);
                    }
                    else {
                        d_rho_particles[tid] = result_i;
                        break;
                    }
                }

                //SPH density estimate by SPHtoGrid
                //Corrects the density estimate for the kernel bias
                //See Dehnen & Aly 2012, eq. 18 + 19
                if (!isinf(d_rho_particles[tid])) {
                    d_rho_particles[tid] = utility::dense::sph_kernel::bias_correction(rho_kernel, d_rho_particles[tid], d_mass_particles[tid], h_inv, k);
                }

            }
            else {
                d_radius_particles[tid] = result;
                d_rho_particles[tid] = result;
            }
        }

        void mpi_cycling(int recvPeer, char* recvPtr, size_t totalBytesRecv, int sendPeer, char* sendPtr, size_t totalBytesSend) 
        {
            // char* recvPtr = reinterpret_cast<char*>(d_tree_recv);
            // char* sendPtr = reinterpret_cast<char*>(d_tree);
            // size_t totalBytesRecv = recvCount * sizeof(*d_tree);
            // size_t totalBytesSend = sendCount * sizeof(*d_tree);

            MPI_Request requests[2];
            size_t offsetRecv = 0;
            size_t offsetSend = 0;

            while (offsetRecv < totalBytesRecv || offsetSend < totalBytesSend) {
                int chunkRecv = static_cast<int>(std::min(totalBytesRecv - offsetRecv, static_cast<size_t>(MAX_MPI_BYTES)));
                int chunkSend = static_cast<int>(std::min(totalBytesSend - offsetSend, static_cast<size_t>(MAX_MPI_BYTES)));

                if (offsetRecv < totalBytesRecv) {
                    CUKD_MPI_CALL(Irecv(recvPtr + offsetRecv, chunkRecv, MPI_BYTE, recvPeer, 0,
                                        MPI_COMM_WORLD, &requests[0]));
                } else {
                    requests[0] = MPI_REQUEST_NULL;
                }

                if (offsetSend < totalBytesSend) {
                    CUKD_MPI_CALL(Isend(sendPtr + offsetSend, chunkSend, MPI_BYTE, sendPeer, 0,
                                        MPI_COMM_WORLD, &requests[1]));
                } else {
                    requests[1] = MPI_REQUEST_NULL;
                }

                CUKD_MPI_CALL(Waitall(2, requests, MPI_STATUSES_IGNORE));

                offsetRecv += chunkRecv;
                offsetSend += chunkSend;
            }

        }

        void run_knn_gpu(float* points, size_t numPointsThatIHave, int k, std::vector<float>& radius_particles, std::vector<float>& rho_particles, std::vector<float>& mass_particles, bool use_cycling, float maxRadius, common::SpaceData::DenseType& rho_kernel)
        {
            int mpi_rank, mpi_size;
            MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
            MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);

            //float maxRadius = std::numeric_limits<float>::infinity();

            // -----------------------------------------------------------------------------
            // find out max num points anybody has, so we can allocate
            // -----------------------------------------------------------------------------
            int N = numPointsThatIHave;
            int maxNumPointsAnybodyHas = 0;

            CUKD_MPI_CALL(Allreduce(&N, &maxNumPointsAnybodyHas, 1, MPI_INT, MPI_MAX, MPI_COMM_WORLD));

            float3* d_tree = 0;
            float3* d_tree_recv = 0;
            //int N = myPoints.size();
            // alloc N+1 so we can store one more if anytoher rank gets oen more point
            CUKD_CUDA_CALL(MallocManaged((void**)&d_tree, (maxNumPointsAnybodyHas + 1) * sizeof(float3)));
            CUKD_CUDA_CALL(MallocManaged((void**)&d_tree_recv, (maxNumPointsAnybodyHas + 1) * sizeof(float3)));
            CUKD_CUDA_CALL(Memcpy(d_tree, points, (size_t)N * sizeof(float3),
                hipMemcpyDefault));


            double start_time, end_time;
            // Start timing before your main computation
            MPI_Barrier(MPI_COMM_WORLD);
            start_time = MPI_Wtime();

            cukd::buildTree(d_tree, N);

            // End timing after computation
            MPI_Barrier(MPI_COMM_WORLD);
            end_time = MPI_Wtime();

            // Print results on rank 0
            if (mpi_rank == 0) {
                printf("Total execution time (buildTree): %.6f seconds\n", end_time - start_time);
            }

            int numQueries = N;// myPoints.size();

#if 0 //nosplit
            float3* d_queries;            
            uint64_t* d_cand;
            CUKD_CUDA_CALL(MallocManaged((void**)&d_queries, (size_t)N * sizeof(float3)));
            CUKD_CUDA_CALL(Memcpy(d_queries, points, N * sizeof(float3), hipMemcpyDefault));
            CUKD_CUDA_CALL(MallocManaged((void**)&d_cand, (size_t)N * k * sizeof(uint64_t)));

#else
            //TODO!!!
            // Choose S (number of splits/batches)
            int S = 1;//k; // or make this a parameter

            if (S > N) {
                S = 1;
            }

            size_t batch_size = numQueries / S;

            // Allocate output buffers for all queries
            radius_particles.resize(numPointsThatIHave);
            rho_particles.resize(numPointsThatIHave);

            //float* d_radius_particles = 0;
            //float* d_rho_particles = 0;
            //float* d_mass_particles = 0;
            //CUKD_CUDA_CALL(MallocManaged((void**)&d_radius_particles, numPointsThatIHave * sizeof(float)));
            //CUKD_CUDA_CALL(MallocManaged((void**)&d_rho_particles, numPointsThatIHave * sizeof(float)));
            //CUKD_CUDA_CALL(MallocManaged((void**)&d_mass_particles, numPointsThatIHave * sizeof(float)));
            //CUKD_CUDA_CALL(Memcpy(d_mass_particles, mass_particles.data(), numPointsThatIHave * sizeof(float), hipMemcpyHostToDevice));
#endif            

            // -----------------------------------------------------------------------------
            // now, do the queries and cycling:
            // -----------------------------------------------------------------------------
            MPI_Barrier(MPI_COMM_WORLD);
            start_time = MPI_Wtime();

			int round_size = mpi_size;
			if (!use_cycling) {
				round_size = 1;
			}

			// Allocate
            uint64_t* d_cand = nullptr;
            size_t d_cand_size = 0;

            float3* d_batch_queries = nullptr;
			size_t d_batch_queries_size = 0;

            float* d_batch_mass = nullptr;
			size_t d_batch_mass_size = 0;

            float* d_batch_radius = nullptr;
			size_t d_batch_radius_size = 0;

            float* d_batch_rho = nullptr;
			size_t d_batch_rho_size = 0;

            for (int round = 0; round < round_size; round++) {
				if (mpi_rank == 0) {
					printf("Starting round (cudakdtree-gpu cycling) %d\n", round);
				}  

                if (round == 0) {
                    // nothing to do , we already have our own tree
                }
                else {
                    MPI_Request requests[2];
                    int sendCount = N;
                    int recvCount = 0;
                    int sendPeer = (mpi_rank + 1) % mpi_size;
                    int recvPeer = (mpi_rank + mpi_size - 1) % mpi_size;
                    CUKD_MPI_CALL(Irecv(&recvCount, 1 * sizeof(int), MPI_BYTE, recvPeer, 0,
                        MPI_COMM_WORLD, &requests[0]));
                    CUKD_MPI_CALL(Isend(&sendCount, 1 * sizeof(int), MPI_BYTE, sendPeer, 0,
                        MPI_COMM_WORLD, &requests[1]));
                    CUKD_MPI_CALL(Waitall(2, requests, MPI_STATUSES_IGNORE));

                    char* recvPtr = reinterpret_cast<char*>(d_tree_recv);
                    char* sendPtr = reinterpret_cast<char*>(d_tree);
                    size_t totalBytesRecv = recvCount * sizeof(*d_tree);
                    size_t totalBytesSend = sendCount * sizeof(*d_tree);
                    mpi_cycling(recvPeer, recvPtr, totalBytesRecv, sendPeer, sendPtr, totalBytesSend);

                    N = recvCount;
                    std::swap(d_tree, d_tree_recv);
                }

#if 0 //nosplit                
                // -----------------------------------------------------------------------------
                runQuery << <divRoundUp(numQueries, 1024), 1024 >> >
                    (/* tree */d_tree, N,
                        /* query params */d_cand, k, maxRadius,
                        /* query points */d_queries, numQueries,
                        round);
                CUKD_CUDA_CALL(DeviceSynchronize());
#else
                // Process queries in S batches
                for (int s = 0; s < S; ++s) {
                    size_t start = s * batch_size;
                    size_t this_batch = (s == S - 1) ? numQueries - start : batch_size;
                    if (this_batch == 0) continue;

                    // Allocate only for this batch
                    if (d_cand_size < this_batch * (size_t)k * sizeof(uint64_t)) {

                        if (d_cand != nullptr) {
							CUKD_CUDA_CALL(Free(d_cand));
                        }

                        d_cand_size = this_batch * (size_t)k * sizeof(uint64_t);
                        CUKD_CUDA_CALL(MallocManaged((void**)&d_cand, d_cand_size));
                    }
					CUKD_CUDA_CALL(Memset(d_cand, 0, d_cand_size));

                    // Prepare batch queries
					if (d_batch_queries_size < this_batch * sizeof(float3)) {
						if (d_batch_queries != nullptr) {
							CUKD_CUDA_CALL(Free(d_batch_queries));
						}
						d_batch_queries_size = this_batch * sizeof(float3);
                        CUKD_CUDA_CALL(MallocManaged((void**)&d_batch_queries, d_batch_queries_size));
					}
                    CUKD_CUDA_CALL(Memcpy(d_batch_queries, ((float3*)points) + start, this_batch * sizeof(float3), hipMemcpyDefault));

                    // Prepare batch mass
					if (d_batch_mass_size < this_batch * sizeof(float)) {
						if (d_batch_mass != nullptr) {
							CUKD_CUDA_CALL(Free(d_batch_mass));
						}
						d_batch_mass_size = this_batch * sizeof(float);
                        CUKD_CUDA_CALL(MallocManaged((void**)&d_batch_mass, d_batch_mass_size));
					}                   
                    CUKD_CUDA_CALL(Memcpy(d_batch_mass, mass_particles.data() + start, this_batch * sizeof(float), hipMemcpyHostToDevice));

                    // Output for this batch
					if (d_batch_radius_size < this_batch * sizeof(float)) {
						if (d_batch_radius != nullptr) {
							CUKD_CUDA_CALL(Free(d_batch_radius));
						}
						d_batch_radius_size = this_batch * sizeof(float);
                        CUKD_CUDA_CALL(MallocManaged((void**)&d_batch_radius, d_batch_radius_size));
					}
					CUKD_CUDA_CALL(Memset(d_batch_radius, 0, d_batch_radius_size));

					if (d_batch_rho_size < this_batch * sizeof(float)) {
						if (d_batch_rho != nullptr) {
							CUKD_CUDA_CALL(Free(d_batch_rho));
						}
						d_batch_rho_size = this_batch * sizeof(float);
                        CUKD_CUDA_CALL(MallocManaged((void**)&d_batch_rho, d_batch_rho_size));
					}
					CUKD_CUDA_CALL(Memset(d_batch_rho, 0, d_batch_rho_size));

                    // Run query for this batch
                    runQuery << <divRoundUp(this_batch, 1024ULL), 1024ULL >> >(
                        d_tree, N,
                        d_cand, k, maxRadius,
                        d_batch_queries, this_batch,
                        round
                    );
                    CUKD_CUDA_CALL(DeviceSynchronize());

                    // Extract results for this batch
                    extractFinalResult<<<divRoundUp(this_batch, 1024ULL), 1024ULL >>>(
						d_batch_radius, d_batch_rho, d_batch_mass, this_batch, k, d_cand, rho_kernel
                    );
                    CUKD_CUDA_CALL(DeviceSynchronize());

                    // Copy results to the full output arrays
                    CUKD_CUDA_CALL(Memcpy(radius_particles.data() + start, d_batch_radius, this_batch * sizeof(float), hipMemcpyDeviceToHost));
                    CUKD_CUDA_CALL(Memcpy(rho_particles.data() + start, d_batch_rho, this_batch * sizeof(float), hipMemcpyDeviceToHost));
                }
#endif
            }

            // Free batch memory
            CUKD_CUDA_CALL(Free(d_cand));
            CUKD_CUDA_CALL(Free(d_batch_queries));
            CUKD_CUDA_CALL(Free(d_batch_mass));
            CUKD_CUDA_CALL(Free(d_batch_radius));
            CUKD_CUDA_CALL(Free(d_batch_rho));

            // End timing after computation
            MPI_Barrier(MPI_COMM_WORLD);
            end_time = MPI_Wtime();

            // Print results on rank 0
            if (mpi_rank == 0) {
                printf("Total execution time (queries and cycling are done): %.6f seconds\n", end_time - start_time);
            }

#if 0  //nosplit            
            std::cout << "done all queries..." << std::endl;
            float* d_radius_particles = 0;
            float* d_rho_particles = 0;
            float* d_mass_particles = 0;
            CUKD_CUDA_CALL(MallocManaged((void**)&d_radius_particles, numPointsThatIHave * sizeof(float)));
            CUKD_CUDA_CALL(MallocManaged((void**)&d_rho_particles, numPointsThatIHave * sizeof(float)));
            
            CUKD_CUDA_CALL(MallocManaged((void**)&d_mass_particles, numPointsThatIHave * sizeof(float)));
            CUKD_CUDA_CALL(Memcpy(d_mass_particles, mass_particles.data(), numPointsThatIHave * sizeof(float), hipMemcpyHostToDevice));

            extractFinalResult << <divRoundUp(numQueries, 1024), 1024 >> >
                (d_radius_particles, d_rho_particles, d_mass_particles, numQueries, k, d_cand);

            CUKD_CUDA_CALL(DeviceSynchronize());

            radius_particles.resize(numPointsThatIHave);
            rho_particles.resize(numPointsThatIHave);
            CUKD_CUDA_CALL(Memcpy(radius_particles.data(), d_radius_particles, numPointsThatIHave * sizeof(float), hipMemcpyDeviceToHost));
            CUKD_CUDA_CALL(Memcpy(rho_particles.data(), d_rho_particles, numPointsThatIHave * sizeof(float), hipMemcpyDeviceToHost));

            MPI_Barrier(MPI_COMM_WORLD);
#endif            
        }

        void runQuery_host(
            float3 *tree, size_t N,
            uint64_t *candidateLists, int k, float maxRadius,
            float3 *queries, size_t numQueries,
            int round)
        {
#pragma omp parallel for
            for (size_t tid = 0; tid < numQueries; tid++) {
                float3 qp = queries[tid];
                cukd::FlexHeapCandidateList cl(&candidateLists[(size_t)k * tid], k,
                    round == 0 ? maxRadius : -1.f);
                cukd::stackFree::knn(cl, qp, tree, N);
            }
        }

        void extractFinalResult_host(
            float *radius_particles,
            float *rho_particles,
            float *mass_particles,
            size_t numPoints,
            int k,
            uint64_t *candidateLists,
            common::SpaceData::DenseType rho_kernel
            )
        {
            //utility::dense::sph_kernel::WendlandC6 kernel_wendland;

#pragma omp parallel for
            for (size_t tid = 0; tid < numPoints; tid++) {
                cukd::FlexHeapCandidateList cl(&candidateLists[(size_t)k * tid], k, -1.f);
                float result = cl.returnValue();
                if (!isinf(result)) {
                    result = sqrtf(result);

                    radius_particles[tid] = result;

                    double h_inv = 1.0 / radius_particles[tid];

                    for (int i = 0; i < k; i++) {
                        float result_i = cukd::uint_as_float(cl.entry[i] >> 32); //cl.get_dist2(i);
                        if (!isinf(result_i) && !isinf(result)) {
                            result_i = sqrtf(result_i);

                            // SPH density estimate by SPHtoGrid						
                            rho_particles[tid] += mass_particles[tid] * utility::dense::sph_kernel::W(rho_kernel, result_i * h_inv, h_inv);
                        }
                        else {
                            rho_particles[tid] = result_i;
                            break;
                        }
                    }

                    //SPH density estimate by SPHtoGrid
                    //Corrects the density estimate for the kernel bias
                    //See Dehnen & Aly 2012, eq. 18 + 19
                    if (!isinf(rho_particles[tid])) {
                        rho_particles[tid] = utility::dense::sph_kernel::bias_correction(rho_kernel, rho_particles[tid], mass_particles[tid], h_inv, k);
                    }
                }
                else {
                    radius_particles[tid] = result;
					rho_particles[tid] = result;
                }
            }
        }

        void run_knn_cpu(float *points, size_t numPointsThatIHave, int k, std::vector<float> &radius_particles, std::vector<float>& rho_particles, std::vector<float>& mass_particles, bool use_cycling, float maxRadius, common::SpaceData::DenseType& rho_kernel)
        {
            int mpi_rank, mpi_size;
            MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
            MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);

            //float maxRadius = std::numeric_limits<float>::infinity();

            // -----------------------------------------------------------------------------
            // find out max num points anybody has, so we can allocate
            // -----------------------------------------------------------------------------
            int N = numPointsThatIHave;
            int maxNumPointsAnybodyHas = 0;

            CUKD_MPI_CALL(Allreduce(&N, &maxNumPointsAnybodyHas, 1, MPI_INT, MPI_MAX, MPI_COMM_WORLD));

            std::vector<float3> tree((maxNumPointsAnybodyHas + 1));
            std::vector<float3> tree_recv((maxNumPointsAnybodyHas + 1));
            memcpy(tree.data(), points, N * sizeof(float3));

            // Add timing to your mpiHugeQuery.cu
            double start_time, end_time;
            // Start timing before your main computation
            MPI_Barrier(MPI_COMM_WORLD);
            start_time = MPI_Wtime();

            cukd::buildTree_host(tree.data(), N);

            // End timing after computation
            MPI_Barrier(MPI_COMM_WORLD);
            end_time = MPI_Wtime();

            // Print results on rank 0
            if (mpi_rank == 0) {
                printf("Total execution time (buildTree_host): %.6f seconds\n", end_time - start_time);
            }

            size_t numQueries = N;

#if 0  //nosplit            
            std::vector<float3>  queries(N);
            memcpy(queries.data(), points, (size_t)N * sizeof(float3));
            std::vector<uint64_t>  cand((size_t)N * k);
#else
            //TODO!!!
            int S = 1;//k; // for example, or make it a parameter

            if (S > N) {
                S = 1;
            }

            size_t batch_size = numQueries / S;

            // Allocate output buffers for all queries
            radius_particles.resize(numPointsThatIHave);
            rho_particles.resize(numPointsThatIHave);            
#endif            

            // -----------------------------------------------------------------------------
            // now, do the queries and cycling:
            // -----------------------------------------------------------------------------
            MPI_Barrier(MPI_COMM_WORLD);
            start_time = MPI_Wtime();

			int round_size = mpi_size;
			if (!use_cycling) {
				round_size = 1;
			}

            std::vector<uint64_t> cand;
            std::vector<float> batch_radius;
            std::vector<float> batch_rho;

            for (int round = 0; round < round_size; round++) {
				if (mpi_rank == 0) {
					printf("Starting round (cudakdtree-cpu cycling) %d\n", round);
				}                

                if (round == 0) {
                    // nothing to do , we already have our own tree
                }
                else {
                    MPI_Request requests[2];
                    int sendCount = N;
                    int recvCount = 0;
                    int sendPeer = (mpi_rank + 1) % mpi_size;
                    int recvPeer = (mpi_rank + mpi_size - 1) % mpi_size;
                    CUKD_MPI_CALL(Irecv(&recvCount, 1 * sizeof(int), MPI_BYTE, recvPeer, 0,
                        MPI_COMM_WORLD, &requests[0]));
                    CUKD_MPI_CALL(Isend(&sendCount, 1 * sizeof(int), MPI_BYTE, sendPeer, 0,
                        MPI_COMM_WORLD, &requests[1]));
                    CUKD_MPI_CALL(Waitall(2, requests, MPI_STATUSES_IGNORE));

                    char* recvPtr = reinterpret_cast<char*>(tree_recv.data());
                    char* sendPtr = reinterpret_cast<char*>(tree.data());
                    size_t totalBytesRecv = recvCount * sizeof(float3);
                    size_t totalBytesSend = sendCount * sizeof(float3);
                    mpi_cycling(recvPeer, recvPtr, totalBytesRecv, sendPeer, sendPtr, totalBytesSend);               

                    N = recvCount;
                    std::swap(tree, tree_recv);
                }
#if 0  //nosplit                
                // -----------------------------------------------------------------------------
                runQuery_host(tree, N,
                    cand, k, maxRadius,
                    queries, numQueries,
                    round);
#else
                // Process queries in S batches
                for (int s = 0; s < S; ++s) {
                    size_t start = s * batch_size;
                    //size_t end = std::min(start + batch_size, numQueries);
                    //size_t this_batch = end - start;
                    size_t this_batch = (s == S - 1) ? numQueries - start : batch_size;

                    if (this_batch == 0) continue;

                    // Allocate only for this batch
                    if (this_batch * (size_t)k > cand.size()) {
                        cand.resize(this_batch * (size_t)k);
                    }
					memset(cand.data(), 0, this_batch * (size_t)k * sizeof(uint64_t));

                    if (this_batch > batch_radius.size()) {
						batch_radius.resize(this_batch);
                    }
					memset(batch_radius.data(), 0, this_batch * sizeof(float));

                    if (this_batch > batch_rho.size()) {
                        batch_rho.resize(this_batch);
                    }
					memset(batch_rho.data(), 0, this_batch * sizeof(float));

                    // Prepare batch queries
                    float3 *batch_queries = (float3*)points + start;

                    // Run query for this batch
                    runQuery_host(tree.data(), N, cand.data(), k, maxRadius, batch_queries, this_batch, round);

                    // Extract results for this batch
                    float *batch_mass = mass_particles.data() + start;

                    extractFinalResult_host(batch_radius.data(), batch_rho.data(), batch_mass, this_batch, k, cand.data(), rho_kernel);

                    // Copy results to the full output arrays
                    std::copy(batch_radius.begin(), batch_radius.end(), radius_particles.begin() + start);
                    std::copy(batch_rho.begin(), batch_rho.end(), rho_particles.begin() + start);
                }
#endif                    
            }

            // End timing after computation
            MPI_Barrier(MPI_COMM_WORLD);
            end_time = MPI_Wtime();

            // Print results on rank 0
            if (mpi_rank == 0) {
                printf("Total execution time (queries and cycling are done): %.6f seconds\n", end_time - start_time);
            }

#if 0  //nosplit
            radius_particles.resize(numPointsThatIHave);
            rho_particles.resize(numPointsThatIHave);
            extractFinalResult_host(radius_particles, rho_particles, mass_particles, numPointsThatIHave, k, cand);

            MPI_Barrier(MPI_COMM_WORLD);
#endif           
        }

        void run_knn(float* points, size_t N, int k, std::vector<float>& radius_particles, std::vector<float>& rho_particles, std::vector<float>& mass_particles, bool use_gpu, bool use_cycling, float max_radius, common::SpaceData::DenseType& rho_kernel)
        {
            if (use_gpu)
                run_knn_gpu(points, N, k, radius_particles, rho_particles, mass_particles, use_cycling, max_radius, rho_kernel);
            else
                run_knn_cpu(points, N, k, radius_particles, rho_particles, mass_particles, use_cycling, max_radius, rho_kernel);
        }

	}// cudakdtree
} //utility